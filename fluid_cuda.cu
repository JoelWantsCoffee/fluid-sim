#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_vector_types.h>

#define flow density

#include "fluid.h"
#include "fluid_cuda.cuh"

void check_error(hipError_t e)
{
    if (e != hipSuccess)
    {
        printf("CUDA error: %d : %s\n", int(e), hipGetErrorString(e));
        abort();
    }
}

__global__ void multiply_into_simple(int N, int stride_a, const float* __restrict__ A, int stride_b, const float* __restrict__ B, int stride_c, float* __restrict__ C)
{
    __shared__ float As[32][32];
    __shared__ float Bs[32][32];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    float lilsum = 0;

    for (int k = 0; k < N; k += 32)
    {
        As[threadIdx.y][threadIdx.x] = A[j*stride_a + k + threadIdx.x];
        Bs[threadIdx.y][threadIdx.x] = B[(k + threadIdx.y)*stride_b + i];
        __syncthreads();

        #pragma unroll
        for (int l = 0; l < 32; l++)
        {
            lilsum += As[threadIdx.y][l] * Bs[l][threadIdx.x];
        }
        __syncthreads();
    }

    C[j*stride_c + i] = lilsum;
}

__global__ void project_block_simple(float * flow, float * from_velx, float * from_vely, float * to_velx, float * to_vely) 
{
    /*
    
    tu = t + 1i
    tv = t + 1j

    float s = t->density + t->density + tu->density + tv->density;
    
    s = !s ? 0 : (1 / s);

    float d = (t->vel_x + t->vel_y - tu->vel_x - tv->vel_y) * s;

    (t - from + into)->vel_x -= d * t->density;
    (t - from + into)->vel_y -= d * t->density;
    (tu - from + into)->vel_x += d * tu->density;
    (tv - from + into)->vel_y += d * tv->density;
    
    */

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    int index = i + j * WIDTH;

    to_velx[index] = from_velx[index];
    to_vely[index] = from_vely[index];
}

__host__ void project_all_gpu(struct Tile * from, struct Tile * to)
{
    // Load memory onto the GPU
    float * flow;    check_error(hipMalloc(&flow, WIDTH*HEIGHT*sizeof(float)));
    float * from_velx;  check_error(hipMalloc(&from_velx, WIDTH*HEIGHT*sizeof(float)));
    float * from_vely;  check_error(hipMalloc(&from_vely, WIDTH*HEIGHT*sizeof(float)));
    float * to_velx;    check_error(hipMalloc(&to_velx, WIDTH*HEIGHT*sizeof(float)));
    float * to_vely;    check_error(hipMalloc(&to_vely, WIDTH*HEIGHT*sizeof(float)));

    for (int j = 0; j < HEIGHT; j++)
    for (int i = 0; i < WIDTH; i++)
    {
        int index = i + j * WIDTH;
        flow[index] = from[index].flow;
        from_velx[index] = from[index].vel_x;
        from_vely[index] = from[index].vel_y;
        to_velx[index] = 0;
        to_vely[index] = 0;
    }

    // Do computation
    project_block_simple<<<dim3(WIDTH / 32, HEIGHT / 32, 1), dim3(32,32,1)>>>(flow, from_velx, from_vely, to_velx, to_vely);
    check_error(hipPeekAtLastError());
    check_error(hipDeviceSynchronize());


    // Load memory back from the GPU
    for (int j = 0; j < HEIGHT; j++)
    for (int i = 0; i < WIDTH; i++)
    {
        int index = i + j * WIDTH;
        to[index].vel_x = to_velx[index];
        to[index].vel_y = to_vely[index];
    }

    // check_error(hipMemcpy(C, C_device, N*N*sizeof(float), hipMemcpyDeviceToHost));

    check_error(hipFree(flow));
    check_error(hipFree(from_velx));
    check_error(hipFree(from_vely));
    check_error(hipFree(to_velx));
    check_error(hipFree(to_vely));
}